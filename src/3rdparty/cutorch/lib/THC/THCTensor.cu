#include "THCTensor.h"

hipTextureObject_t THCudaTensor_getTextureObject(THCState *state, THCudaTensor *self)
{
  THAssert(THCudaTensor_checkGPU(state, 1, self));
  hipTextureObject_t texObj;
  struct hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeLinear;
  resDesc.res.linear.devPtr = THCudaTensor_data(state, self);
  resDesc.res.linear.sizeInBytes = THCudaTensor_nElement(state, self) * 4;
  resDesc.res.linear.desc = hipCreateChannelDesc(32, 0, 0, 0,
                                                  hipChannelFormatKindFloat);
  struct hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);
  hipError_t errcode = hipGetLastError();
  if(errcode != hipSuccess) {
    if (THCudaTensor_nElement(state, self) > 2>>27)
      THError("Failed to create texture object, "
              "nElement:%ld exceeds 27-bit addressing required for tex1Dfetch. Cuda Error: %s",
              THCudaTensor_nElement(state, self), hipGetErrorString(errcode));
    else
      THError("Failed to create texture object: %s", hipGetErrorString(errcode));
  }
  return texObj;
}

THC_API int THCudaTensor_getDevice(THCState* state, const THCudaTensor* thc) {
  if (!thc->storage) return -1;
  hipPointerAttribute_t attr;
  THCudaCheck(hipPointerGetAttributes(&attr, thc->storage->data));
  return attr.device;
}
