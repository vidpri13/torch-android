#include "THCStorageCopy.h"
#include "THCGeneral.h"

void THCudaStorage_rawCopy(THCState *state, THCudaStorage *self, float *src)
{
  THCudaCheck(hipMemcpyAsync(self->data, src, self->size * sizeof(float), hipMemcpyDeviceToDevice, THCState_getCurrentStream(state)));
}

void THCudaStorage_copy(THCState *state, THCudaStorage *self, THCudaStorage *src)
{
  THArgCheck(self->size == src->size, 2, "size does not match");
  THCudaCheck(hipMemcpyAsync(self->data, src->data, self->size * sizeof(float), hipMemcpyDeviceToDevice, THCState_getCurrentStream(state)));
}

void THCudaStorage_copyCuda(THCState *state, THCudaStorage *self, THCudaStorage *src)
{
  THArgCheck(self->size == src->size, 2, "size does not match");
  THCudaCheck(hipMemcpyAsync(self->data, src->data, self->size * sizeof(float), hipMemcpyDeviceToDevice, THCState_getCurrentStream(state)));
}
