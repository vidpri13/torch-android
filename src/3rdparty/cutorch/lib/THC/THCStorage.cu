#include "THCStorage.h"

#include <thrust/device_ptr.h>
#include <thrust/fill.h>
#if CUDA_VERSION >= 7000
#include <thrust/system/cuda/execution_policy.h>
#endif

void THCudaStorage_fill(THCState *state, THCudaStorage *self, float value)
{
  thrust::device_ptr<float> self_data(self->data);
  thrust::fill(
#if CUDA_VERSION >= 7000
    thrust::cuda::par.on(THCState_getCurrentStream(state)),
#endif
    self_data, self_data+self->size, value);
}

void THCudaStorage_resize(THCState *state, THCudaStorage *self, long size)
{
  THArgCheck(size >= 0, 2, "invalid size");

  if(!(self->flag & TH_STORAGE_RESIZABLE))
    return;

  if(size == 0)
  {
    if(self->flag & TH_STORAGE_FREEMEM) {
      THCudaCheck(THCudaFree(state, self->data));
      THCHeapUpdate(state, -self->size * sizeof(float));
    }
    self->data = NULL;
    self->size = 0;
  }
  else
  {
    float *data = NULL;
    // update heap *before* attempting malloc, to free space for the malloc
    THCHeapUpdate(state, size * sizeof(float));
    hipError_t err = THCudaMalloc(state, (void**)(&data), size * sizeof(float));
    if(err != hipSuccess) {
      THCHeapUpdate(state, -size * sizeof(float));
    }
    THCudaCheck(err);

    if (self->data) {
      THCudaCheck(hipMemcpyAsync(data,
                                  self->data,
                                  THMin(self->size, size) * sizeof(float),
                                  hipMemcpyDeviceToDevice,
                                  THCState_getCurrentStream(state)));
      THCudaCheck(THCudaFree(state, self->data));
      THCHeapUpdate(state, -self->size * sizeof(float));
    }

    self->data = data;
    self->size = size;
  }
}
